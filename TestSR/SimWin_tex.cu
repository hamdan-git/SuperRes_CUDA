#include "hip/hip_runtime.h"

#include "Kernel.h"

#include <stdio.h>
#include <math.h>
#include <float.h>


// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex_img;
texture<unsigned char, 2, hipReadModeElementType> tex_map;

//-------------------------------------------------------------------
__global__ void SimWin_Kernel_tex(float *outputImagekernel, int imageWidth, int imageHeight)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	int iOffset = row * imageWidth + col;

	if (tex2D(tex_map, col, row) == 0)
	{
		outputImagekernel[iOffset] = tex2D(tex_img, col, row);// inputImageKernel[iOffset];
		return;
	}
	//return;
	const int iLargeWin = 6;
	const int iWin = 2;


	const int iWinSize = (2 * iWin + 1) * (2 * iWin + 1);

	float pCurValues[iWinSize];
	for (int i = 0; i < iWinSize; i++) 
		pCurValues[i] = -1.0f;

	float fValidSum = 0.0;
	int iIndex = 0;
	int iCurValidNeighbours = 0;
	for (int j = -iWin; j <= iWin; j++)
	{
		for (int i = -iWin; i <= iWin; i++)
		{
			int iNewX = i + col;
			int iNewY = j + row;
			if (iNewX >= 0 && iNewX < imageWidth && iNewY >= 0 && iNewY < imageHeight)
			{
				unsigned char iLabel = tex2D(tex_map, iNewX, iNewY);// ata[iOffset + (j*imageWidth) + i];
				if (iLabel == 0)
				{
					//pCurValues[iIndex] = inputImageKernel[iOffset + (j*imageWidth) + i];
					pCurValues[iIndex] = tex2D(tex_img, iNewX, iNewY);
					iCurValidNeighbours++;
					fValidSum += pCurValues[iIndex];
				}
			}
			iIndex++;
		}
	}



	float fMinDiff = FLT_MAX;

	float fCurentWinMean = 1.0f;
	if (iCurValidNeighbours > 0)
		fCurentWinMean = fValidSum / (double)iCurValidNeighbours;
	int iHalfCurValidNeighbours = (iCurValidNeighbours >> 1) + 1; //added on 10Aug11

	int iStartX = col - iLargeWin + iWin;
	if (iStartX < iWin) iStartX = iWin;
	int iStartY = row - iLargeWin + iWin;
	if (iStartY < iWin) iStartY = iWin;
	int iEndX = iLargeWin + col - iWin;
	if (iEndX >= imageWidth - iWin) iEndX = imageWidth - iWin - 1;
	int iEndY = iLargeWin + row - iWin;
	if (iEndY >= imageHeight - iWin) iEndY = imageHeight - iWin - 1;

	int iTargetX = -1, iTargetY = -1;

	//float *pLocOutData = &outputImagekernel[iStartY*imageWidth + iStartX];
	//unsigned char *pLocMapData = &pMapData[iStartY*imageWidth + iStartX];
	int iJumpX = imageWidth - (iEndX - iStartX + 1);
	for (int jj = iStartY; jj <= iEndY; jj++)
	{
		for (int ii = iStartX; ii <= iEndX; ii++)
		{
			if ((ii != col || jj != row) && (tex2D(tex_map, ii, jj) == 0 )) //ha added on 25Feb2015, cluster size larger than the large win size were not update w/out the last term
			{
				//now search the local win 
				int iIndex = 0;
				int iLocalValidNeighbours = 0;
				double fDiffSum = 0.0;
				//double fSum = 0;
				for (int j = -iWin; j <= iWin; j++)
				{
					for (int i = -iWin; i <= iWin; i++)
					{
						int iNewX = ii + i;
						int iNewY = jj + j;
						if (iNewX >= 0 && iNewX < imageWidth && iNewY >= 0 && iNewY < imageHeight && pCurValues[iIndex] >= 0.0)
						{
							int iLabel = tex2D(tex_map, iNewX, iNewY); ;// *(pLocMapData + (j*imageWidth + i));
							if (iLabel == 0 /*|| iLabel == iLocalLabel*/)
							{
								iLocalValidNeighbours++;
								float pNeiVal = tex2D(tex_img, iNewX,  iNewY);
								double fDiff = fabs((double)(pNeiVal - pCurValues[iIndex]));
								fDiffSum += fDiff;

							}
						}
						iIndex++;
					}
				}
				if (iLocalValidNeighbours >= iHalfCurValidNeighbours && iLocalValidNeighbours > 0) //added on 10Aug11
				{
					fDiffSum /= (double)iLocalValidNeighbours;
					if (fDiffSum < fMinDiff)
					{
						fMinDiff = fDiffSum;
						iTargetX = ii;
						iTargetY = jj;

					}
				}


			}
			//pLocMapData++;
			//pLocOutData++;

		}
		//pLocMapData += iJumpX;
		//pLocOutData += iJumpX;
	}
	if (iTargetX >= 0 && iTargetY >= 0)
	{
		//outputImagekernel[iOffset] = inputImageKernel[iTargetY*imageWidth + iTargetX];// (pOrigData[iTargetY*iWidth + iTargetX]);
		outputImagekernel[iOffset] = tex2D(tex_img, iTargetX, iTargetY);// +iTargetX];// (pOrigData[iTargetY*iWidth + iTargetX]);
	}

}



//--------------------------------------------------------------------------------
hipError_t SimWin_tex_Cuda(float *pInData, float *pOutData, int iWidth, int iHeight, int iNumFrames, unsigned char *pMapData)
{

	//float *d_InData = 0;
	hipArray *cuArray_img;
	hipArray *cuArray_map;
	float *d_OutData = 0;

	int iFrameSize = iWidth * iHeight;

	hipError_t cudaStatus;



	// Choose which GPU to run on, change this on a multi-GPU system.
	//cudaStatus = hipSetDevice(0);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	//	goto Error;
	//}

// Allocate array and copy image data

	//imput image text array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMallocArray(&cuArray_img, &channelDesc, iWidth, iHeight);
	//hipMemcpyToArray(cuArray_img, 0, 0, pInData, iFrameSize*sizeof(float), hipMemcpyHostToDevice);
	hipBindTextureToArray(tex_img, cuArray_img, channelDesc);

	//defectmap text array
	hipChannelFormatDesc channelDesc_map = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipMallocArray(&cuArray_map, &channelDesc_map, iWidth, iHeight);
	hipMemcpyToArray(cuArray_map, 0, 0, pMapData, iFrameSize , hipMemcpyHostToDevice);
	hipBindTextureToArray(tex_map, cuArray_map, channelDesc_map);

	cudaStatus = hipMalloc((void**)&d_OutData, iFrameSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}



	int TILE_SIZE_X = 1;
	int TILE_SIZE_Y = 128;
	dim3 dimBlock(TILE_SIZE_X, TILE_SIZE_Y);

	dim3 dimGrid((int)ceil((float)iWidth / (float)TILE_SIZE_X), (int)ceil((float)iHeight / (float)TILE_SIZE_Y));

	for (int z = 0; z < iNumFrames; z++)
	{
		hipMemcpyToArray(cuArray_img, 0, 0, &pInData[z*iFrameSize], iFrameSize * sizeof(float), hipMemcpyHostToDevice);

		// Launch a kernel on the GPU with one thread for each element.
		SimWin_Kernel_tex << <dimGrid, dimBlock >> > (d_OutData, iWidth, iHeight);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(&pOutData[z*iFrameSize], d_OutData, iFrameSize * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	}

Error:
	hipFreeArray(cuArray_img);
	hipFreeArray(cuArray_map);
	hipFree(d_OutData);

	return cudaStatus;
}


