#include "hip/hip_runtime.h"
#include "Kernel.h"

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "IPTools.h"
#include <math.h>
#include <algorithm>
#include "Common.h"

#ifndef min
#define min(a,b)            (((a) < (b)) ? (a) : (b))
#endif
#ifndef max
#define max(a,b)            (((a) > (b)) ? (a) : (b))
#endif

// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex_rot_imgA;


//-----------------------------------------------------------------------------
__device__ double getInterpolatedPixelA(double x, double y, int iWidth, int iHeight, float* pixels)
{
	int xbase = (int)x;
	int ybase = (int)y;
	xbase = xbase < iWidth ? xbase : iWidth - 1;// min(xbase, iWidth - 1);
	ybase = ybase < iHeight ? ybase : iHeight - 1;// (ybase, iHeight - 1);
	//if (xbase  >= iWidth  ybase >= iHeight )
	//	return 1;
	double xFraction = x - xbase;
	double yFraction = y - ybase;
	int offset = ybase * iWidth + xbase;
	double lowerLeft = pixels[offset];
	double lowerRight = xbase == iWidth - 1 ? pixels[offset] : pixels[offset + 1];
	double upperRight = (xbase == iWidth - 1 || ybase == iHeight - 1) ? pixels[offset] : pixels[offset + iWidth + 1];
	double upperLeft = ybase == iHeight - 1 ? pixels[offset] : pixels[offset + iWidth];
	double upperAverage = upperLeft;
	if (xFraction != 0.0)
		upperAverage += xFraction * (upperRight - upperLeft);
	double lowerAverage = lowerLeft;
	if (xFraction != 0.0)
		lowerAverage += xFraction * (lowerRight - lowerLeft);
	if (yFraction == 0.0)
		return lowerAverage;
	else
		return lowerAverage + yFraction * (upperAverage - lowerAverage);
}

//-------------------------------------------------------------------
__global__ void RotateImageA_tex_kernel(float *outputImagekernel, int inWidth, int inHeight, int outWidth, int outHeight, double theta)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float rads = (theta) * 3.1415926 / 180.0;

	float u = (float)col - (float)outWidth / 2;
	float v = (float)row - (float)outHeight / 2;
	float tu = u * cosf(rads) - v * sinf(rads);
	float tv = v * cosf(rads) + u * sinf(rads);

	tu /= (float)inWidth;
	tv /= (float)inHeight;



	if (col < outWidth && row < outHeight)
		outputImagekernel[row*outWidth + col] = tex2D(tex_rot_imgA, tu + 0.5f, tv + 0.5f);
}

//-------------------------------------------------------------------
__global__ void RotateImageA_kernel(float *inImagekernel, float *outputImagekernel, int inWidth, int inHeight, int outWidth, int outHeight, double theta, double fMag)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float rads = (theta) * 3.1415926 / 180.0;

	float u = (float)col - (float)outWidth / 2;
	float v = (float)row - (float)outHeight / 2;
	float tu = u * cosf(rads) - v * sinf(rads);
	float tv = v * cosf(rads) + u * sinf(rads);

	//tu /= (float)inWidth;
	//tv /= (float)inHeight;
	tu *= fMag;
	tv *= fMag;
	tu = (float)(tu + (float)inWidth / 2.0f);
	tv = (float)(tv + (float)inHeight / 2.0f);


	if (/*col < outWidth && row < outHeight &&*/ tu >= 0 && tu < inWidth && tv >= 0 && tv < inHeight)
	{
		outputImagekernel[row*outWidth + col] = inImagekernel[((int)tv * inWidth) + (int)tu];//tex2D(tex_rot_imgA, tu + inWidth/2 , tv + inHeight/2);
		//outputImagekernel[row*outWidth + col] = getInterpolatedPixelA(tu, tv, inWidth, inHeight, inImagekernel);
	}
}

//---------------------------
__global__ void RotateImageA_usingLUT_kernel(float *inImagekernel, float *outputImagekernel, int inWidth, int inHeight, int outWidth, int outHeight, double theta, int *pLut)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int offset = row * outWidth + col;
	if (col < outWidth && row < outHeight )
	{
		if ( pLut[offset] >= 0 )
			outputImagekernel[offset] = inImagekernel[pLut[offset]];
	}
}


__global__ void SmoothBorder_kernel(float *inImagekernel, float *outputImagekernel, unsigned char *pMaskData, int iWidth, int iHeight, int iWin)
{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int iOffset = row * iWidth + col;
	outputImagekernel[iOffset] = inImagekernel[iOffset] * inImagekernel[iOffset];
	//if (pMaskData[iOffset] > 0)
	//{
	//	int count = 0;
	//	double fSum = 0.0;
	//	for (int j = -iWin; j <= iWin; j++)
	//	{
	//		for (int i = -iWin; i <= iWin; i++)
	//		{
	//			int iNewX = col + i;
	//			int iNewY = row + j;
	//			if (iNewX >= 0 && iNewX < iWidth && iNewY >= 0 && iNewY < iHeight)
	//			{
	//				fSum += (double)inImagekernel[iNewY*iWidth + iNewX];
	//				count++;
	//			}
	//		}
	//	}
	//	if (count > 0)
	//	{
	//		outputImagekernel[iOffset] = (float)(fSum / (double)count);
	//	}
	//}
}
//-------------------------------------------------------------------
__global__ void TDS_AddA_kernel(float *inFrameData,  float *outputImageData, int inWidth, int inHeight)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int offset = row * inWidth + col;
	if ( col < inWidth && row < inHeight)
		outputImageData[offset] += inFrameData[offset];

	//__syncthreads();
}

//-------------------------------------------------------------------
__global__ void TDS_AddA_conditional_kernel(float *inFrameData, float *outputImageData, int inWidth, int inHeight)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float pVals[2];
	int offset = row * inWidth + col;

	if (col < inWidth && row < inHeight)
	{
		float fVal = inFrameData[offset];
		if ( fVal > 0.0f)
			outputImageData[offset] += fVal;
	}
}

//-------------------------------------------------------------------
__global__ void SetValues_kernel(float *inFrameData, float iVal, int inWidth, int inHeight)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < inWidth && row < inHeight)
		inFrameData[row*inWidth + col] = iVal;
}

//-------------------------------------------------------------------
__global__ void Memcpy_us_to_float__kernel(unsigned short *pInData, float *pOutData, int inWidth, int inHeight)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < inWidth && row < inHeight)
		pOutData[row*inWidth + col] = pInData[row*inWidth + col];
}

//-------------------------------------------------------------------
__global__ void Memcpy_any_to_float__kernel(unsigned char *pInData, float *pOutData, int inWidth, int inHeight, int pixType)

{
	// Set row and colum for thread.
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < inWidth && row < inHeight)
	{
		switch (pixType)
		{
		case 1:
			pOutData[row*inWidth + col] = (float)*((unsigned char*)&pInData[row*inWidth + col]);
			break;
		case 2:
			pOutData[row*inWidth + col] = (float)*((unsigned short*)&pInData[row*inWidth*2 + col*2]);
			break;
		case 4:
			pOutData[row*inWidth + col] = (float)*((unsigned int*)&pInData[row*inWidth*4 + col*4]);
			break;
		case 6:
			pOutData[row*inWidth + col] = (float)*((float*)&pInData[row*inWidth*4 + col*4]);
			break;

		};
	}
}

//----------------------------------------------------------------------------
unsigned char *GetMaskDataAfterRotation(int iW, int iH, double theta, double fMag, int &iNewW, int &iNewH)
{
	unsigned char *pMaskData = NULL;
	//int iNewW=0, iNewH = 0;
	FindDimensionAfterRotation(iW, iH, theta, fMag, iNewW, iNewH);
	if (iNewW <= 0 || iNewH <= 0) return NULL;
	pMaskData = new unsigned char[iNewW*iNewH];
	unsigned char *pTempData = new unsigned char[iW*iH];
	unsigned char *pRotatedTempData = new unsigned char[iNewW*iNewH];
	memset(pRotatedTempData, 0, iNewW*iNewH);

	for (long i = 0; i < iW*iH; i++)
		pTempData[i] = 1;

	unsigned char *pMaks_rot_eroded = new unsigned char[iNewW*iNewH];
	memset(pMaks_rot_eroded, 0, iNewW*iNewH);
	IPTools<unsigned char>::RotateImage_cpu(pTempData, iW, iH, pRotatedTempData, iNewW, iNewH, theta, 0);
	IPTools<unsigned char>::DoErosion(pRotatedTempData, pMaks_rot_eroded, iNewW, iNewH, 5);
	float *pDisData = new float[iNewW*iNewH];
	IPTools<unsigned char>::GetDistanceMap(pMaks_rot_eroded, iNewW, iNewH, 0, pDisData);

	//WriteRawData<unsigned char>("c:\\Temp\\Dist.raw", pMaks_rot_eroded, iNewW, iNewH);

	memset(pMaskData, 0, 1 * iNewW*iNewH);
	for (long i = 0; i < iNewW*iNewH; i++)
	{
		float fVal = pDisData[i];
		if (fVal > 0.0f && fVal < 6.0f)
			pMaskData[i] = 1;
	}

	delete[] pTempData;
	delete[] pRotatedTempData;
	delete[] pMaks_rot_eroded;
	delete[] pDisData;
	return pMaskData;
}

//--------------------------------------------------------------------
void RotateImage_GetLUT_cpu(int iW, int iH, int *pLut, int iOutWidth, int iOutHeight, double theta, double fMagnification)
{
	for (long i = 0; i < iOutWidth * iOutHeight; i++) pLut[i] = -1;
	long iInFrameSize = iW * iH;
	double fMag = 1.0 / fMagnification;
	float rads = (theta) * 3.1415926 / 180.0;
	float cs = cos(rads); // precalculate these values
	float ss = sin(rads);
	float xcenterOut = (float)(iOutWidth) / 2.0;   // use float here!
	float ycenterOut = (float)(iOutHeight) / 2.0;
	float xcenterIn = (float)iW / 2.0f;
	float ycenterIn = (float)iH / 2.0f;
	for (int row = 0; row < iOutHeight; row++)
	{
		for (int col = 0; col < iOutWidth; col++)
		{
			float u = (float)col - xcenterOut;
			float v = (float)row - ycenterOut;
			float tu = u * cs - v * ss;
			float tv = v * cs + u * ss;

			tu *= fMag;
			tv *= fMag;
			tu += xcenterIn;
			tv += ycenterIn;
			//tu += (iOutWidth - iW) / 2;
			//tu += (iOutHeight - iH) / 2;

			if (tu >= 0 && tu < iW && tv >= 0 && tv < iH)
			{
				//pOutData[row*iOutWidth + col] = getInterpolatedPixel_TF(tu, tv, iW, iH,  pData);
				long offset = (int)tv*iW + (int)tu;
				// pLut[offset] = row*iOutWidth + col;
				pLut[row*iOutWidth + col] = offset;
			}
		}
	}
}


//--------------------------------------------------------------------------------
//hipError_t RotateAddImage_Cuda(unsigned short* pInData, int inWidth, int inHeight, int iNumFrames, unsigned short *pOutData, int outWidth, int outHeight, double theta, double fScale, double fMag)
//{
//
//	hipArray *cuArray_img;
//	float *d_OutData = 0;
//	unsigned short *d_InData_us = 0;
//	float *d_InData = 0;
//	float *d_RotatedFrameData = 0;
//	unsigned char *d_pMaskData = 0;
//	float *pTempInData = 0;
//	float *pTempOutData = 0;
//	int *pLut = 0;
//	int *d_pLut = 0;
//	unsigned char *pMaskData = NULL;
//
//	int iFrameSize = inWidth * inHeight;
//	int iOutFrameSize = outWidth * outHeight;
//
//	pTempInData = new float[iFrameSize];
//	pTempOutData = new float[iOutFrameSize];
//
//
//	hipError_t cudaStatus = hipErrorInvalidValue;
//
//	int iRotWidth, int iRotHeight;
//	FindDimensionAfterRotation(inWidth, inHeight, theta, fMag, iRotWidth, iRotHeight);
//
//	int iRotatedFrameSize = iRotWidth * iRotHeight;
//
//	pLut = new int[iRotatedFrameSize];
//	RotateImage_GetLUT_cpu(inWidth, inHeight, pLut, iRotWidth, iRotHeight, theta, fMag);
//
//	//Get mask data to cover arround the edges after rotation
//	int iNewMaskW = 0, iNewMaskH = 0;
//	pMaskData = GetMaskDataAfterRotation(inWidth, inHeight, theta, fMag, iNewMaskW, iNewMaskH);
//	if (iNewMaskW != iRotWidth || iNewMaskH != iRotHeight)
//	{
//		printf("dimension mismatch when creating mask\n");
//		goto Error;
//	}
//	//WriteRawData<unsigned char>("c:\\Temp\\MaskData.raw", pMaskData, iNewMaskW, iNewMaskH);
//
//	printf("Rotaed dim %d %d %d %d\n", iRotWidth, iRotHeight, outWidth,  outHeight);
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	//cudaStatus = hipSetDevice(0);
//	//if (cudaStatus != hipSuccess) {
//	//          fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//	//          goto Error;
//	//}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	//imput image text array
////	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
////	hipMallocArray(&cuArray_img, &channelDesc, inWidth, inHeight);
//////	hipMemcpyToArray(cuArray_img, 0, 0, pInData, iFrameSize * sizeof(unsigned short), hipMemcpyHostToDevice);
////	// Set texture parameters
////	tex_rot_imgA.addressMode[0] = hipAddressModeBorder;// ModeWrap;
////	tex_rot_imgA.addressMode[1] = hipAddressModeBorder;
////	tex_rot_imgA.filterMode = hipFilterModeLinear;
////	tex_rot_imgA.normalized = true;    // access with normalized texture coordinates
////	hipBindTextureToArray(tex_rot_imgA, cuArray_img, channelDesc);
//
//
//
//	cudaStatus = hipMalloc((void**)&d_OutData, iOutFrameSize * sizeof(float));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&d_InData_us, iFrameSize * sizeof(unsigned short));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&d_InData, iFrameSize * sizeof(float));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&d_RotatedFrameData, iRotatedFrameSize * sizeof(float)*2);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//		
//	cudaStatus = hipMalloc((void**)&d_pLut, iRotatedFrameSize * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//	hipMemcpy(d_pLut, pLut, iRotatedFrameSize * sizeof(int), hipMemcpyHostToDevice);
//
//	cudaStatus = hipMalloc((void**)&d_pMaskData, iRotatedFrameSize * sizeof(unsigned char) );
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
////	hipMemcpy(d_pMaskData, pMaskData, iRotatedFrameSize * sizeof(unsigned char), hipMemcpyHostToDevice);
//
//	int TILE_SIZE_X = 16;
//	int TILE_SIZE_Y = 16;
//	dim3 dimBlock(TILE_SIZE_X, TILE_SIZE_Y);
//
//	dim3 dimGrid((int)ceil((float)iRotWidth / (float)TILE_SIZE_X), (int)ceil((float)iRotHeight / (float)TILE_SIZE_Y));
//	
//	dim3 dimGrid_in((int)ceil((float)inWidth / (float)TILE_SIZE_X), (int)ceil((float)inHeight / (float)TILE_SIZE_Y));
//
//	dim3 dimGrid_out((int)ceil((float)outWidth / (float)TILE_SIZE_X), (int)ceil((float)outHeight / (float)TILE_SIZE_Y));
//
//	double fShiftRow = 0.0;
//	bool bReversed = false;
//	int iCurIndex = 0;
//	int iPrevIndex = -1;
//	SetValues_kernel << <dimGrid_out, dimBlock >> > (d_OutData, 0, outWidth, outHeight);
//
//
//	for (int iZ = 0; iZ < iNumFrames /*&&iZ<30*/ ; iZ++)
//	{
//		iCurIndex = (int)fShiftRow*outWidth;
//		if (iPrevIndex != iCurIndex)
//		{
//			
//
//			int iZIndex = bReversed ? iZ : iNumFrames - 1 - iZ;
//
//			//////////////////
//			//unsigned short *pInDataRef = &pInData[iZIndex * iFrameSize];
//			//for (int k = 0; k < iFrameSize; k++)
//			//	pTempInData[k] = (float)pInDataRef[k];
//			//hipMemcpy(d_InData, pTempInData, iFrameSize * sizeof(float), hipMemcpyHostToDevice);
//			///////////////////// 
//
//			hipMemcpy(d_InData_us, &pInData[iZIndex * iFrameSize], iFrameSize * sizeof(unsigned short), hipMemcpyHostToDevice);
//			Memcpy_us_to_float__kernel << <dimGrid_in, dimBlock>> > (d_InData_us, d_InData, inWidth, inHeight);
//
//
//			SetValues_kernel << <dimGrid, dimBlock >> > (d_RotatedFrameData, 0, iRotWidth, iRotHeight);
//			RotateImageA_kernel << <dimGrid, dimBlock >> > (d_InData, &d_RotatedFrameData[0], inWidth, inHeight, iRotWidth, iRotHeight, theta, 1.0 / fMag);
//
//
//
//		//	RotateImageA_usingLUT_kernel << <dimGrid, dimBlock >> > (d_InData, &d_RotatedFrameData[0], inWidth, inHeight, iRotWidth, iRotHeight, theta, d_pLut);
//			//SmoothBorder_kernel << <dimGrid, dimBlock >> > (&d_RotatedFrameData[0], &d_RotatedFrameData[iRotatedFrameSize], d_pMaskData, iRotWidth, iRotHeight, 3);
//			TDS_AddA_kernel << <dimGrid, dimBlock >> > (&d_RotatedFrameData[0], &d_OutData[iCurIndex], iRotWidth, iRotHeight);
//			
//			hipDeviceSynchronize();
//		}
//		fShiftRow += fScale;
//		if (fShiftRow >= outHeight) break;
//		iPrevIndex = iCurIndex;
//	}
//
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//	
//	// Copy output vector from GPU buffer to host memory.
//	//cudaStatus = hipMemcpy(pOutData, d_OutData, iOutFrameSize * sizeof(unsigned short), hipMemcpyDeviceToHost);
//	cudaStatus = hipMemcpy(pTempOutData, d_OutData, iOutFrameSize * sizeof(float), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//	for (int k = 0; k < iOutFrameSize; k++)
//		pOutData[k] = (unsigned short)pTempOutData[k];
//	
//Error:
//	//hipFreeArray(cuArray_img);
//	if ( d_InData!=NULL )hipFree(d_InData);
//	if (d_OutData!=NULL) hipFree(d_OutData);
//	if (d_RotatedFrameData!=NULL) hipFree(d_RotatedFrameData);
//	if (d_pMaskData!=NULL)hipFree(d_pMaskData);
//	if (d_InData_us != NULL)hipFree(d_InData_us);
//	if (d_pLut != NULL) hipFree(d_pLut);
//	delete[] pMaskData;
//	delete[] pTempInData;
//	delete[] pTempOutData;
//	delete[] pLut;
//	return cudaStatus;
//}

//--------------------------------------------------------------------------------------------------------------------------
hipError_t RotateAddImage_Cuda(unsigned char* pInData, int inWidth, int inHeight, int iNumFrames, unsigned char *pOutData, int outWidth, int outHeight, int pixType, double theta, double fScale, double fMag, bool bReversed)
{

	hipArray *cuArray_img;
	float *d_OutData = 0;
	unsigned char *d_InData_any = 0;
	float *d_InData = 0;
	float *d_RotatedFrameData = 0;
	unsigned char *d_pMaskData = 0;
	float *pTempInData = 0;
	float *pTempOutData = 0;
	int *pLut = 0;
	int *d_pLut = 0;
	unsigned char *pMaskData = NULL;

	int iFrameSize = inWidth * inHeight;
	int iOutFrameSize = outWidth * outHeight;

	pTempInData = new float[iFrameSize];
	pTempOutData = new float[iOutFrameSize];

	int pixSize = 2; //u16
	switch (pixType)
	{
	case 1: pixSize = 1;
	case 4: case 6: pixSize = 4; break; //u32 and float

	};


	hipError_t cudaStatus = hipErrorInvalidValue;

	int iRotWidth, int iRotHeight;
	FindDimensionAfterRotation(inWidth, inHeight, theta, fMag, iRotWidth, iRotHeight);

	int iRotatedFrameSize = iRotWidth * iRotHeight;

	pLut = new int[iRotatedFrameSize];
	RotateImage_GetLUT_cpu(inWidth, inHeight, pLut, iRotWidth, iRotHeight, theta, fMag);

	//Get mask data to cover arround the edges after rotation
	int iNewMaskW = 0, iNewMaskH = 0;
	pMaskData = GetMaskDataAfterRotation(inWidth, inHeight, theta, fMag, iNewMaskW, iNewMaskH);
	if (iNewMaskW != iRotWidth || iNewMaskH != iRotHeight)
	{
		printf("dimension mismatch when creating mask\n");
		goto Error;
	}
	//WriteRawData<unsigned char>("c:\\Temp\\MaskData.raw", pMaskData, iNewMaskW, iNewMaskH);

	printf("Rotaed dim %d %d %d %d\n", iRotWidth, iRotHeight, outWidth, outHeight);

	cudaStatus = hipMalloc((void**)&d_OutData, iOutFrameSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_InData_any, iFrameSize * pixSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_InData, iFrameSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_RotatedFrameData, iRotatedFrameSize * sizeof(float) * 2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_pLut, iRotatedFrameSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	hipMemcpy(d_pLut, pLut, iRotatedFrameSize * sizeof(int), hipMemcpyHostToDevice);

	cudaStatus = hipMalloc((void**)&d_pMaskData, iRotatedFrameSize * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//	hipMemcpy(d_pMaskData, pMaskData, iRotatedFrameSize * sizeof(unsigned char), hipMemcpyHostToDevice);

	int TILE_SIZE_X = 16;
	int TILE_SIZE_Y = 16;
	dim3 dimBlock(TILE_SIZE_X, TILE_SIZE_Y);

	dim3 dimGrid((int)ceil((float)iRotWidth / (float)TILE_SIZE_X), (int)ceil((float)iRotHeight / (float)TILE_SIZE_Y));

	dim3 dimGrid_in((int)ceil((float)inWidth / (float)TILE_SIZE_X), (int)ceil((float)inHeight / (float)TILE_SIZE_Y));

	dim3 dimGrid_out((int)ceil((float)outWidth / (float)TILE_SIZE_X), (int)ceil((float)outHeight / (float)TILE_SIZE_Y));

	float fShiftRow = 0.0;
	//bool bReversed = false;
	int iCurIndex = 0;
	int iPrevIndex = -1;
	SetValues_kernel << <dimGrid_out, dimBlock >> > (d_OutData, 0, outWidth, outHeight);


	for (int iZ = 0; iZ < iNumFrames /*&&iZ<30*/; iZ++)
	{
		iCurIndex = (int)fShiftRow*outWidth;
		if (iPrevIndex != iCurIndex)
		{


			int iZIndex = bReversed ? iZ : iNumFrames - 1 - iZ;

			//////////////////
			//unsigned short *pInDataRef = &pInData[iZIndex * iFrameSize];
			//for (int k = 0; k < iFrameSize; k++)
			//	pTempInData[k] = (float)pInDataRef[k];
			//hipMemcpy(d_InData, pTempInData, iFrameSize * sizeof(float), hipMemcpyHostToDevice);
			///////////////////// 

			hipMemcpy(d_InData_any, &pInData[iZIndex * iFrameSize * pixSize], iFrameSize * pixSize, hipMemcpyHostToDevice);
			Memcpy_any_to_float__kernel << <dimGrid_in, dimBlock >> > (d_InData_any, d_InData, inWidth, inHeight, pixType);
			//Memcpy_us_to_float__kernel << <dimGrid_in, dimBlock >> > ((unsigned short*)d_InData_any, d_InData, inWidth, inHeight);

			SetValues_kernel << <dimGrid, dimBlock >> > (d_RotatedFrameData, 0, iRotWidth, iRotHeight);
			RotateImageA_kernel << <dimGrid, dimBlock >> > (d_InData, &d_RotatedFrameData[0], inWidth, inHeight, iRotWidth, iRotHeight, theta, 1.0 / fMag);
			//RotateImageA_usingLUT_kernel << <dimGrid, dimBlock >> > (d_InData, &d_RotatedFrameData[0], inWidth, inHeight, iRotWidth, iRotHeight, theta, d_pLut);
			TDS_AddA_kernel << <dimGrid, dimBlock >> > (&d_RotatedFrameData[0], &d_OutData[iCurIndex], iRotWidth, iRotHeight);

			hipDeviceSynchronize();
		}
		fShiftRow += fScale;
		if (fShiftRow >= outHeight) break;
		iPrevIndex = iCurIndex;
	}
	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy(pOutData, d_OutData, iOutFrameSize * sizeof(unsigned short), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(pTempOutData, d_OutData, iOutFrameSize * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	switch (pixType)
	{
	case 1: for (int k = 0, int p = 0; p < iOutFrameSize; k += pixSize, p++) *((unsigned char*)&pOutData[k]) = (unsigned char)pTempOutData[p]; break;
	case 2: for (int k = 0, int p = 0; p < iOutFrameSize; k += pixSize, p++) *((unsigned short*)&pOutData[k]) = (unsigned short)pTempOutData[p]; break;
	case 4: for (int k = 0, int p = 0; p < iOutFrameSize; k += pixSize, p++) pOutData[k] = (unsigned int)pTempOutData[p]; break;
	case 6: for (int k = 0, int p = 0; p < iOutFrameSize; k += pixSize, p++) pOutData[k] = (float)pTempOutData[p]; break;
	};

Error:
	//hipFreeArray(cuArray_img);
	if (d_InData != NULL)hipFree(d_InData);
	if (d_OutData != NULL) hipFree(d_OutData);
	if (d_RotatedFrameData != NULL) hipFree(d_RotatedFrameData);
	if (d_pMaskData != NULL)hipFree(d_pMaskData);
	if (d_InData_any != NULL)hipFree(d_InData_any);
	if (d_pLut != NULL) hipFree(d_pLut);
	delete[] pMaskData;
	delete[] pTempInData;
	delete[] pTempOutData;
	delete[] pLut;
	return cudaStatus;
}
//--------------------------------------------------------------------------------
hipError_t RotateAddImage_lut_Cuda(unsigned char* pInData, int inWidth, int inHeight, int iNumFrames, unsigned char *pOutData, int outWidth, int outHeight, int pixType, double theta, double fScale, double fMag, bool bReversed)
{

	float *d_OutData = 0;
	unsigned char *d_InData_any = 0;
	float *d_InData = 0;
	float *d_RotatedFrameData = 0;
	float *pTempInData = 0;
	float *pTempOutData = 0;
	int *pLut = 0;
	int *d_pLut = 0;
	unsigned char *pMaskData = NULL;


	int iFrameSize = inWidth * inHeight;
	int iOutFrameSize = outWidth * outHeight;

	pTempInData = new float[iFrameSize];
	pTempOutData = new float[iOutFrameSize];

	int pixSize = 2; //u16
	switch (pixType)
	{
	case 1: pixSize = 1;
	case 4: case 6: pixSize = 4; break; //u32 and float

	};

	hipError_t cudaStatus = hipErrorInvalidValue;

	int iRotWidth, int iRotHeight;
	FindDimensionAfterRotation(inWidth, inHeight, theta, fMag, iRotWidth, iRotHeight);

	int iRotatedFrameSize = iRotWidth * iRotHeight;

	pLut = new int[iRotatedFrameSize];
	RotateImage_GetLUT_cpu(inWidth, inHeight, pLut, iRotWidth, iRotHeight, theta, fMag);



	printf("Rotaed dim %d %d %d %d\n", iRotWidth, iRotHeight, outWidth, outHeight);

	cudaStatus = hipMalloc((void**)&d_OutData, iOutFrameSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_InData_any, iFrameSize * pixSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_InData, iFrameSize * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_RotatedFrameData, iRotatedFrameSize * sizeof(float) * 2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_pLut, iRotatedFrameSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	hipMemcpy(d_pLut, pLut, iRotatedFrameSize * sizeof(int), hipMemcpyHostToDevice);



	int TILE_SIZE_X = 32;
	int TILE_SIZE_Y = 8;
	dim3 dimBlock(TILE_SIZE_X, TILE_SIZE_Y);

	dim3 dimGrid((int)ceil((float)iRotWidth / (float)TILE_SIZE_X), (int)ceil((float)iRotHeight / (float)TILE_SIZE_Y));

	dim3 dimGrid_in((int)ceil((float)inWidth / (float)TILE_SIZE_X), (int)ceil((float)inHeight / (float)TILE_SIZE_Y));

	float fShiftRow = 0.0f;
	//bool bReversed = false;
	//bReversed = false;
	int iCurIndex = 0;
	int iPrevIndex = -1;
	for (int iZ = 0; iZ < iNumFrames; iZ++)
	{
		iCurIndex = (int)fShiftRow*outWidth;
		if (iPrevIndex != iCurIndex)
		{
			int iZIndex = bReversed ? iZ : iNumFrames - 1 - iZ;

			//////////////////
			//unsigned short *pInDataRef = &pInData[iZIndex * iFrameSize];
			//for (int k = 0; k < iFrameSize; k++)
			//	pTempInData[k] = (float)pInDataRef[k];
			//hipMemcpy(d_InData, pTempInData, iFrameSize * sizeof(float), hipMemcpyHostToDevice);
			///////////////////// 

			hipMemcpy(d_InData_any, &pInData[iZIndex * iFrameSize * pixSize], iFrameSize * pixSize, hipMemcpyHostToDevice);
			Memcpy_any_to_float__kernel << <dimGrid_in, dimBlock >> > (d_InData_any, d_InData, inWidth, inHeight, pixType);
			//Memcpy_us_to_float__kernel << <dimGrid_in, dimBlock >> > ((unsigned short*)d_InData_any, d_InData, inWidth, inHeight);

			SetValues_kernel << <dimGrid, dimBlock >> > (d_RotatedFrameData, 0, iRotWidth, iRotHeight);
			//RotateImageA_kernel << <dimGrid, dimBlock >> > (d_InData, &d_RotatedFrameData[0], inWidth, inHeight, iRotWidth, iRotHeight, theta, 1.0 / fMag);
			RotateImageA_usingLUT_kernel << <dimGrid, dimBlock >> > (d_InData, &d_RotatedFrameData[0], inWidth, inHeight, iRotWidth, iRotHeight, theta, d_pLut);
//			TDS_AddA_kernel << <dimGrid, dimBlock >> > (&d_RotatedFrameData[0], &d_OutData[iCurIndex], iRotWidth, iRotHeight);
			TDS_AddA_conditional_kernel << <dimGrid, dimBlock >> > (&d_RotatedFrameData[0], &d_OutData[iCurIndex], iRotWidth, iRotHeight);

			hipDeviceSynchronize();
		}
		fShiftRow += fScale;
		iPrevIndex = iCurIndex;
		if (fShiftRow >= outHeight) break;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy(pOutData, d_OutData, iOutFrameSize * sizeof(unsigned short), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(pTempOutData, d_OutData, iOutFrameSize * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	/////////////////////
	//FILE *out; fopen_s(&out, "c:\\temp\\testOut.raw", "wb"); fwrite(pTempOutData, 4, iOutFrameSize, out); fclose(out);
	////////////////////
	switch (pixType)
	{
	case 1: for (int k = 0, int p=0; p < iOutFrameSize; k+=pixSize, p++) *((unsigned char*)&pOutData[k]) = (unsigned char)pTempOutData[p]; break;
	case 2: for (int k = 0, int p = 0; p < iOutFrameSize; k += pixSize, p++) *((unsigned short*)&pOutData[k]) = (unsigned short)pTempOutData[p]; break;
	case 4: for (int k = 0, int p = 0; p < iOutFrameSize; k += pixSize, p++) pOutData[k] = (unsigned int)pTempOutData[p]; break;
	case 6: for (int k = 0, int p = 0; p < iOutFrameSize; k += pixSize, p++) pOutData[k] = (float)pTempOutData[p]; break;
	};


Error:
	//hipFreeArray(cuArray_img);
	if (d_InData != NULL)hipFree(d_InData);
	if (d_OutData != NULL) hipFree(d_OutData);
	if (d_RotatedFrameData != NULL) hipFree(d_RotatedFrameData);
	if (d_InData_any != NULL)hipFree(d_InData_any);
	if (d_pLut != NULL) hipFree(d_pLut);
	delete[] pMaskData;
	delete[] pTempInData;
	delete[] pTempOutData;
	delete[] pLut;
	return cudaStatus;
}
